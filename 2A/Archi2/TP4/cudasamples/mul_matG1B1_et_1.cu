 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
 

// CUDA Kernel
__global__ void matrixMul( float* C, float* A, float* B, int TM)
{
   float cc;
   int k;
   // calcul des coordonnees du thread
   int i =  blockIdx.x;
   int j =  threadIdx.x;
 
   cc = 0.;
   // calcul de c[i][j]
   for(k=0;k<TM;k++){
      cc += A[i*TM+k] * B[k*TM+j];
   }
   C[i*TM+j] = cc;
}
 
 
/////////////////////////////////////////////////////////
// Programme main
/////////////////////////////////////////////////////////
 
int main(int argc, char** argv)
{
	int i, j, GRID_SIZE_X, BLOCK_SIZE_X;
	int TM=1024;
   int size = TM*TM*sizeof(float);
	hipError_t cerror;

	if (argc>1) {
		 TM=atoi(argv[1]);
	}
	BLOCK_SIZE_X = TM; 
	GRID_SIZE_X = TM;
 
   // definiton de la grille et des blocs
   dim3 block(BLOCK_SIZE_X);
   dim3 grid(GRID_SIZE_X);

   // allocation des matrices sur CPU
   float *h_A, *h_B, *h_C ;

   h_A = (float*) malloc(size);
   h_B = (float*) malloc(size);
   h_C = (float*) malloc(size);


   
   // initialisation des matrices avec des valeurs permettant de verifier le resultat
   for(i = 0; i < TM; i++){
      for(j = 0; j < TM; j++){
      	h_A[i*TM+j] = 1.0;
      	h_B[i*TM+j] = 1.0;
		h_C[i*TM+j] = 0.0;

		if (i==j) {
			h_A[i*TM+j]=(float) (i+1);
			h_B[i*TM+j]=(float) (i+1);
		}
      }
   }

   // Pour mesurer le temps de calcul
   float tc, tt_cpuGpu, tt_gpuCpu;
   hipEvent_t depart, arret,cpuGpuDebut, cpuGpuFin, gpuCpuDebut, gpuCpuFin;
   hipEventCreate(&depart);
   hipEventCreate(&arret);
   hipEventCreate(&cpuGpuDebut);
   hipEventCreate(&cpuGpuFin);
   hipEventCreate(&gpuCpuDebut);
   hipEventCreate(&gpuCpuFin);
   hipEventRecord(depart,0);
   hipEventRecord(cpuGpuDebut,0);
   hipEventRecord(gpuCpuDebut,0);

   // allocation des matrices sur GPU
   float *d_A, *d_B, *d_C;
   hipMalloc((void**) &d_A,size);
   hipMalloc((void**) &d_B,size);
   hipMalloc((void**) &d_C,size);
 
   // copie des matrives A et B depuis le CPU vers le GPU
   hipMemcpy(d_A, h_A, size,hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size,hipMemcpyHostToDevice);
      // Mesure du temps : transfert + calcul
   hipEventRecord(cpuGpuFin,0);
   hipEventSynchronize(cpuGpuFin);
   hipEventElapsedTime(&tt_cpuGpu,cpuGpuDebut, cpuGpuFin);

   hipEventDestroy(cpuGpuDebut);
   hipEventDestroy(cpuGpuFin);
   printf("Transfert A,B de CPU vers GPU : %f seconde\n", tt_cpuGpu/1000.0);

   // lancement des threads
   matrixMul<<< grid, block >>>(d_C, d_A, d_B, TM);
 
   // Vérification en cas d'erreur
   cerror=hipGetLastError(); 
   if ((int)cerror !=0) { 
     printf("Erreur appel kernel %d \n", (int) cerror);
     exit(cerror);
   }
   // copie de la matrive C depuis le GPU
   hipMemcpy(h_C,d_C, size,hipMemcpyDeviceToHost);
   // Mesure du temps : transfert + calcul
   hipEventRecord(gpuCpuFin,0);
   hipEventSynchronize(gpuCpuFin);
   hipEventElapsedTime(&tt_gpuCpu,gpuCpuDebut, gpuCpuFin);

   hipEventDestroy(gpuCpuDebut);
   hipEventDestroy(gpuCpuFin);
   printf("Transfert C de GPU vers CPU : %f seconde\n", tt_gpuCpu/1000.0);


   // Mesure du temps : transfert + calcul
   hipEventRecord(arret,0);
   hipEventSynchronize(arret);
   hipEventElapsedTime(&tc,depart, arret);

   hipEventDestroy(depart);
   hipEventDestroy(arret);
   printf("Temps consommé : %f seconde\n", tc/1000.0);

   // verification du resultat
  for(i = 0; i < TM; i++){
    for(j = 0; j < TM; j++){
	if ((i==j) && (h_C[i*TM+j] != (float)((i+1)*(i+1)+TM-1))) 	   	{
		printf("Erreur i: %d j: %d %f\n", i, j, h_C[i*TM+j] ); exit(1);
		}
		else if ((i!=j) && (h_C[i*TM+j] != (float)(i+j+TM))) 			{
			printf("Erreur i: %d j: %d\n", i, j);
			exit(1);
		}
      }
   } 

   // liberation de la memoire
   free(h_A);
   free(h_B);
   free(h_C);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
 
}
