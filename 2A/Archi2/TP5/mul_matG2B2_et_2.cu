 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
 

#define TILE_SIZE 16



// CUDA Kernel
__global__ void matrixMul( float* C, float* A, float* B, int TM)
{
  __shared__ float As [TILE_SIZE][TILE_SIZE];
  __shared__ float Bs [TILE_SIZE][TILE_SIZE];

    // chaque thread calcule C[i][j]
    // Coordonnees absolues du thread : indices i j
    int j = blockIdx.x * blockDim.x+ threadIdx.x;
    int i = blockIdx.y * blockDim.y+ threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
 
    float value = 0;

    for(int ke=0; ke<TM; ke += TILE_SIZE) {
      // Remplissage de As et de Bs
      As[ty][tx] = A[i * TM + ke + tx];
      Bs[ty][tx] = B[(ke + ty) * TM + j];

      __syncthreads();

      // Calcul
      for (int k = 0; k < TILE_SIZE; k++) {
        value += As[ty][k] * Bs[k][tx];
      }

      __syncthreads();
    }

   C[i * TM + j] = value;
}
  
/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////
 
int main(int argc, char** argv) {
  int i, j, TM, BLOCK_SIZE_X, BLOCK_SIZE_Y;
  unsigned int M_size;
  float *h_A, *h_B, *h_C;
  float *d_A, *d_B, *d_C;
  hipError_t cerror;
  float  elapsedTime ;
  hipEvent_t start , stop ;

  // Valeurs par defaut
  TM=2048;
  BLOCK_SIZE_X = TILE_SIZE;
  BLOCK_SIZE_Y = TILE_SIZE;

  // Possibilite de lire TM dans arg1, BLOCK_SIZE_X dans arg2 et BLOCK_SIZE_Y ans arg3
  if (argc>1) {
    TM=atoi(argv[1]);
  }
  // if (argc>3) {
  //   BLOCK_SIZE_X =atoi(argv[2]);
  //   BLOCK_SIZE_Y =atoi(argv[3]);
  // }
  
  // Verification de la bonne taille TM par rapport aux dimensions des blocs
  if ((TM % BLOCK_SIZE_X) !=0) {
    printf("Taille matrice non multiple de taille bloc X %d \n", BLOCK_SIZE_X);
    exit(1);
  }
  if ((TM % BLOCK_SIZE_Y) !=0) {
    printf("Taille matrice non multiple de taille bloc Y %d \n", BLOCK_SIZE_Y);
    exit(1);
  }
  
  // Allocation memoire sur CPU
  M_size = TM*TM*sizeof(float);
  h_A = (float*) malloc(M_size);
  h_B = (float*) malloc(M_size);
  h_C = (float*) malloc(M_size);
  
  // initialisation des matrices avec des valeurs permettant de verifier le resultat
  for(i = 0; i < TM; i++){
      for(j = 0; j < TM; j++){
      	h_A[i*TM+j] = 1.0;
      	h_B[i*TM+j] = 1.0;
	      h_C[i*TM+j] = 0.0;

      	if (i==j) {
      		h_A[i*TM+j]=(float) (i+1);
      		h_B[i*TM+j]=(float) (i+1);
      	}
      }
  }
  
  // Allocation memoire sur GPU
  hipMalloc((void**) &d_A, M_size);
  hipMalloc((void**) &d_B, M_size);
  hipMalloc((void**) &d_C, M_size);
  
  // Calcul du temps : top depart
  hipEventCreate (&start ) ;
  hipEventCreate (&stop ) ;
  hipEventRecord ( start , 0 ) ;
  
  // copie des donnes CPU vers GPU
  hipMemcpy(d_A, h_A, M_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, M_size, hipMemcpyHostToDevice);
  
  // choix de la structure : grille et blocs
  dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
  dim3 grid(TM / threads.x, TM / threads.y);
 
   printf("bloc %d %d \n", BLOCK_SIZE_X, BLOCK_SIZE_Y);
   printf("grille %d %d \n", TM / threads.x, TM / threads.y);
   
   // Lancement des threads
   matrixMul<<< grid, threads >>>(d_C, d_A, d_B, TM);

   // En cas d'erreur
   cerror=hipGetLastError(); 
   if ((int)cerror !=0) { 
     printf("Erreur appel kernel %d \n", (int) cerror);
     exit(cerror);
   }
 
   // copie des resultats depuis le GPU
   hipMemcpy(h_C, d_C, M_size, hipMemcpyDeviceToHost);
   
   // Calcul du temps d'execution
   hipEventRecord ( stop , 0 ) ;
   hipEventSynchronize ( stop ) ;
   hipEventElapsedTime ( &elapsedTime , start , stop ) ;
   hipEventDestroy ( start ) ;
   hipEventDestroy ( stop ) ;
   printf ( "Temps consomme: %f secondes\n" , elapsedTime / 1000.0 ) ;

  // Verification des resultats

   for(i = 0; i < TM; i++){
     for(j = 0; j < TM; j++){
      	if ((i==j) && (h_C[i*TM+j] != (float)((i+1)*(i+1)+TM-1))) {
      		printf("Erreur i: %d j: %d %f\n", i, j, h_C[i*TM+j] ); exit(1);
      	}
      	else if ((i!=j) && (h_C[i*TM+j] != (float)(i+j+TM))){
      		printf("Erreur i: %d j: %d\n", i, j);
      		exit(1);
      	}
     }
   } 

   // liberation de la memoire
   free(h_A);
   free(h_B);
   free(h_C);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
}
