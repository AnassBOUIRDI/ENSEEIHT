#include "hip/hip_runtime.h"
 
#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
 
// CUDA Kernel
__global__ void matrixMul( float* C, float* A, float* B, int TM)
{
   float cc;
   int k;
   // calcul des coordonnees du thread
   int i =  blockIdx.x;
   int j =  threadIdx.x;
 
   cc = 0.;
   // calcul de c[i][j]
   for(k=0;k<TM;k++){
      cc += A[i*TM+k] * B[k*TM+j];
   }
   C[i*TM+j] = cc;
}
/* ____________________________________________________*/

__global__ void matrixMul( float* C, float* A, float* B, int TM)
{
   // chaque thread calcule C[i][j]
   // Coordonnees absolues du thread : indices i j
   int j = blockIdx.x * blockDim.x+ threadIdx.x;
   int i = blockIdx.y * blockDim.y+ threadIdx.y;
 
   float value = 0;
   for (int k = 0; k < TM; ++k)
   {
      value += A[i * TM + k] * B[k * TM + j];
   }

   C[i * TM + j] = value;
}
/* ____________________________________________________*/

/* ________________ DEROULEMENT BOUCLE FOR (8 best)___*/
 
__global__ void matrixMul( float* C, float* A, float* B, int TM)
{
   // chaque thread calcule C[i][j]
   // Coordonnees absolues du thread : indices i j
   int j = blockIdx.x * blockDim.x+ threadIdx.x;
   int i = blockIdx.y * blockDim.y+ threadIdx.y;
 
   float value = 0;
   for (int k = 0; k < TM; k+= 8)
   {
      value += A[i * TM + k] * B[k * TM + j];
      value += A[i * TM + k+1] * B[(k+1) * TM + j];
      value += A[i * TM + k+2] * B[(k+2) * TM + j];
      value += A[i * TM + k+3] * B[(k+3) * TM + j];
      value += A[i * TM + k+4] * B[(k+4) * TM + j];
      value += A[i * TM + k+5] * B[(k+5) * TM + j];
      value += A[i * TM + k+6] * B[(k+6) * TM + j];
      value += A[i * TM + k+7] * B[(k+7) * TM + j];
   //    value += A[i * TM + k+8] * B[(k+8) * TM + j];
   //    value += A[i * TM + k+9] * B[(k+9) * TM + j];
   //    value += A[i * TM + k+10] * B[(k+10) * TM + j];
   //    value += A[i * TM + k+11] * B[(k+11) * TM + j];
   //    value += A[i * TM + k+12] * B[(k+12) * TM + j];
   //    value += A[i * TM + k+13] * B[(k+13) * TM + j];
   //    value += A[i * TM + k+14] * B[(k+14) * TM + j];
   //    value += A[i * TM + k+15] * B[(k+15) * TM + j];
    }

   C[i * TM + j] = value;
}
/* _____________FIN DEROULEMENT BOUCLE FOR (8 best)___*/

/* ________ UTILISATION MATRICE (=TUILE) PARTAGEE ___*/

__global__ void matrixMul( float* C, float* A, float* B, int TM)
{
  __shared__ float As [TILE_SIZE][TILE_SIZE];
  __shared__ float Bs [TILE_SIZE][TILE_SIZE];

    // chaque thread calcule C[i][j]
    // Coordonnees absolues du thread : indices i j
    int j = blockIdx.x * blockDim.x+ threadIdx.x;
    int i = blockIdx.y * blockDim.y+ threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
 
    float value = 0;

    for(int ke=0; ke<TM; ke += TILE_SIZE) {
      // Remplissage de As et de Bs
      As[ty][tx] = A[i * TM + ke + tx];
      Bs[ty][tx] = B[(ke + ty) * TM + j];

      __syncthreads();

      // Calcul
      for (int k = 0; k < TILE_SIZE; k++) {
        value += As[ty][k] * Bs[k][tx];
      }

      __syncthreads();
    }

   C[i * TM + j] = value;
}

/* _____FIN UTILISATION MATRICE (=TUILE) PARTAGEE ___*/

/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////
 
int main(int argc, char** argv) {
  int i, j, TM, BLOCK_SIZE_X, BLOCK_SIZE_Y;
  unsigned int M_size;
  float *h_A, *h_B, *h_C;
  float *d_A, *d_B, *d_C;
  hipError_t cerror;
  float  elapsedTime ;
  hipEvent_t start , stop ;

  // Valeurs par defaut
  TM=2048;
  BLOCK_SIZE_X = 32; // = TILE SIZE SI TUILE
  BLOCK_SIZE_Y = 32; // = TILE SIZE SI TUILE
 
  // Possibilite de lire TM dans arg1, BLOCK_SIZE_X dans arg2 et BLOCK_SIZE_Y ans arg3
  if (argc>1) {
    TM=atoi(argv[1]);
  }
  if (argc>3) {
    BLOCK_SIZE_X =atoi(argv[2]);
    BLOCK_SIZE_Y =atoi(argv[3]);
  }
  
  // Verification de la bonne taille TM par rapport aux dimensions des blocs
  if ((TM % BLOCK_SIZE_X) !=0) {
    printf("Taille matrice non multiple de taille bloc X %d \n", BLOCK_SIZE_X);
    exit(1);
  }
  if ((TM % BLOCK_SIZE_Y) !=0) {
    printf("Taille matrice non multiple de taille bloc Y %d \n", BLOCK_SIZE_Y);
    exit(1);
  }
  
  // Allocation memoire sur CPU
  M_size = TM*TM*sizeof(float);
  h_A = (float*) malloc(M_size);
  h_B = (float*) malloc(M_size);
  h_C = (float*) malloc(M_size);
  
  // initialisation des matrices avec des valeurs permettant de verifier le resultat
  for(i = 0; i < TM; i++){
      for(j = 0; j < TM; j++){
      	h_A[i*TM+j] = 1.0;
      	h_B[i*TM+j] = 1.0;
	h_C[i*TM+j] = 0.0;

	if (i==j) {
		h_A[i*TM+j]=(float) (i+1);
		h_B[i*TM+j]=(float) (i+1);
	}
      }
  }
  
  // Allocation memoire sur GPU
  hipMalloc((void**) &d_A, M_size);
  hipMalloc((void**) &d_B, M_size);
  hipMalloc((void**) &d_C, M_size);
  
  // Calcul du temps : top depart
  hipEventCreate (&start ) ;
  hipEventCreate (&stop ) ;
  hipEventRecord ( start , 0 ) ;
  
  // copie des donnes CPU vers GPU
  hipMemcpy(d_A, h_A, M_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, M_size, hipMemcpyHostToDevice);
  
  // choix de la structure : grille et blocs
  dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
  dim3 grid(TM / threads.x, TM / threads.y);
 
   printf("bloc %d %d \n", BLOCK_SIZE_X, BLOCK_SIZE_Y);
   printf("grille %d %d \n", TM / threads.x, TM / threads.y);
   
   // Lancement des threads
   matrixMul<<< grid, threads >>>(d_C, d_A, d_B, TM);

   // En cas d'erreur
   cerror=hipGetLastError(); 
   if ((int)cerror !=0) { 
     printf("Erreur appel kernel %d \n", (int) cerror);
     exit(cerror);
   }
 
   // copie des resultats depuis le GPU
   hipMemcpy(h_C, d_C, M_size, hipMemcpyDeviceToHost);
   
   // Calcul du temps d'execution
   hipEventRecord ( stop , 0 ) ;
   hipEventSynchronize ( stop ) ;
   hipEventElapsedTime ( &elapsedTime , start , stop ) ;
   hipEventDestroy ( start ) ;
   hipEventDestroy ( stop ) ;
   printf ( "Temps consomme: %f secondes\n" , elapsedTime / 1000.0 ) ;

  // Verification des resultats

   for(i = 0; i < TM; i++){
     for(j = 0; j < TM; j++){
	if ((i==j) && (h_C[i*TM+j] != (float)((i+1)*(i+1)+TM-1))) {
		printf("Erreur i: %d j: %d %f\n", i, j, h_C[i*TM+j] ); exit(1);
	}
	else if ((i!=j) && (h_C[i*TM+j] != (float)(i+j+TM))){
		printf("Erreur i: %d j: %d\n", i, j);
		exit(1);
	}
     }
   } 

   // liberation de la memoire
   free(h_A);
   free(h_B);
   free(h_C);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
}

Les threads sont exécutés sur le GPU en fonction de leur organisation en blocs :
- chaque thread dispose d'un certain nombre de registres,
- les threads d'un même bloc se partagent un bloc mémoire plus rapide que la mémoire globale,
- les threads d'un même bloc sont exécutés par petits ensembles (warp) de 32 (ou 64 selon les
versions des GPU). Ces warps sont exécutés de façon concurrente et se partagent les mêmes
ressources (unités de calcul, accès mémoire, etc.), et peuvent bénéficier d'un certain parallélisme
lorsqu'ils font appel à des ressources différentes à un instant donné.

Le déroulage boucle (duplication du code à l'intérieur d'une boucle) peut augmenter les
possibilités d'entrelacement des instructions, diminuer les blocages, et améliorer le parallélisme
entre les threads du même bloc.
